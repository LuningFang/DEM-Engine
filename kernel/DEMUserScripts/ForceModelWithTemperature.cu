#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////
// The first part is just the standard full Hertzian--Mindlin
/////////////////////////////////////////////////////////////

// No need to do any contact force calculation if no contact. And it can happen,
// since we added extra contact margin for adding electrostatic forces before
// physical contacts emerge.
/////////////////////////////////////////////////////////////
// The first part is just the standard full Hertzian--Mindlin
/////////////////////////////////////////////////////////////

// No need to do any contact force calculation if no contact. And it can happen,
// since we added extra contact margin for adding electrostatic forces before
// physical contacts emerge.
float E_cnt, G_cnt, CoR_cnt, mu_cnt, Crr_cnt;
const float ks = 2e5;

if (overlapDepth > 0) {
    // Material properties
    // float E_cnt, G_cnt, CoR_cnt, mu_cnt, Crr_cnt;
    {
        // E and nu are associated with each material, so obtain them this way
        float E_A = E[bodyAMatType];
        float nu_A = nu[bodyAMatType];
        float E_B = E[bodyBMatType];
        float nu_B = nu[bodyBMatType];
        matProxy2ContactParam<float>(E_cnt, G_cnt, E_A, nu_A, E_B, nu_B);
        // CoR, mu and Crr are pair-wise, so obtain them this way
        CoR_cnt = CoR[bodyAMatType][bodyBMatType];
        mu_cnt = mu[bodyAMatType][bodyBMatType];
        Crr_cnt = Crr[bodyAMatType][bodyBMatType];
    }

    float3 rotVelCPA, rotVelCPB;
        // We also need the relative velocity between A and B in global frame to use in the damping terms
        // To get that, we need contact points' rotational velocity in GLOBAL frame
        // This is local rotational velocity (the portion of linear vel contributed by rotation)
        rotVelCPA = cross(ARotVel, locCPA);
        rotVelCPB = cross(BRotVel, locCPB);
        // This is mapping from local rotational velocity to global
        applyOriQToVector3<float, deme::oriQ_t>(rotVelCPA.x, rotVelCPA.y, rotVelCPA.z, AOriQ.w, AOriQ.x, AOriQ.y,
                                                AOriQ.z);
        applyOriQToVector3<float, deme::oriQ_t>(rotVelCPB.x, rotVelCPB.y, rotVelCPB.z, BOriQ.w, BOriQ.x, BOriQ.y,
                                                BOriQ.z);

    // A few re-usables
    float mass_eff, sqrt_Rd, beta;
    float3 vrel_tan;
    float3 delta_tan = make_float3(delta_tan_x, delta_tan_y, delta_tan_z);

    // Normal force part
        // The (total) relative linear velocity of A relative to B
        const float3 velB2A = (ALinVel + rotVelCPA) - (BLinVel + rotVelCPB);
        const float projection = dot(velB2A, B2A);
        vrel_tan = velB2A - projection * B2A;

        // Now we already have sufficient info to update contact history
        {
            delta_tan += ts * vrel_tan;
            const float disp_proj = dot(delta_tan, B2A);
            delta_tan -= disp_proj * B2A;
            delta_time += ts;
        }

        mass_eff = (AOwnerMass * BOwnerMass) / (AOwnerMass + BOwnerMass);
        sqrt_Rd = sqrt(overlapDepth * (ARadius * BRadius) / (ARadius + BRadius));
        const float Sn = 2. * E_cnt * sqrt_Rd;

        const float loge = (CoR_cnt < DEME_TINY_FLOAT) ? log(DEME_TINY_FLOAT) : log(CoR_cnt);
        beta = loge / sqrt(loge * loge + deme::PI_SQUARED);

        const float k_n = deme::TWO_OVER_THREE * Sn;
        const float gamma_n = deme::TWO_TIMES_SQRT_FIVE_OVER_SIX * beta * sqrt(Sn * mass_eff);

        force += (k_n * overlapDepth + gamma_n * projection) * B2A;

    // Rolling resistance part

    if (mu_cnt > 0.0) {
        const float kt = 8. * G_cnt * sqrt_Rd;
        const float gt = -deme::TWO_TIMES_SQRT_FIVE_OVER_SIX * beta * sqrt(mass_eff * kt);
        float3 tangent_force = -kt * delta_tan - gt * vrel_tan;
        const float ft = length(tangent_force);
        if (ft > DEME_TINY_FLOAT) {
            // Reverse-engineer to get tangential displacement
            const float ft_max = length(force) * mu_cnt;
            if (ft > ft_max) {
                tangent_force = (ft_max / ft) * tangent_force;
                delta_tan = (tangent_force + gt * vrel_tan) / (-kt);
            }
        } else {
            tangent_force = make_float3(0, 0, 0);
        }
        // Use force to collect tangent_force
        force += tangent_force;
    }

    // Finally, make sure we update those wildcards (in this case, contact history)
    delta_tan_x = delta_tan.x;
    delta_tan_y = delta_tan.y;
    delta_tan_z = delta_tan.z;



    // force magnitude
    double force_mag =  k_n * overlapDepth + gamma_n * projection;

    if (force_mag < 0) {
        force_mag = 0;
    }

    int curr_step = (int)(time / ts);

    if (curr_step % 2000 == 0) {
        // radius contact
        double radius_eff = (ARadius * BRadius) / (ARadius + BRadius);
        double radius_contact = powf( 2.f * radius_eff / E_cnt * force_mag, 1.0/3.0);

        // temperature
        double T_j = Temp_B[BGeo];
        double T_i = Temp_A[AGeo];
        double Q_ij = 2. * ks * radius_contact *  (T_j - T_i);
        atomicAdd(Q_A + AGeo, Q_ij);
        atomicAdd(Q_B + BGeo, -Q_ij);

    }




} else {
    // This is to be more rigorous. If in fact no physical contact, then contact wildcards (such as contact history)
    // should be cleared (they will also be automatically cleared if the contact is no longer detected).
    delta_time = 0;
    delta_tan_x = 0;
    delta_tan_y = 0;
    delta_tan_z = 0;
    
    }

    // this is where particle fluid particle heat exchange model should be added
    int curr_step = (int) (time / ts);
    if (curr_step % 2000 == 0) {
        double distances[11] = {-0.200, -0.160, -0.120, -0.080, -0.040, 0.000, 0.040, 0.080, 0.120, 0.16, 0.200};
        double volumes[11] = {0.150, 0.167, 0.182, 0.197, 0.217, 0.409, 0.077, 0.046, 0.027, 0.012, 0.000};

        double interval = 0.04;
        double start_distance = -0.2;

        // calculate index
        double ratio = overlapDepth / (ARadius + BRadius);
        int i = (int) ((ratio - start_distance) / interval);

        // Check if the distance is within bounds
        if (i > 0 && i < 11 - 1) {

        double t = (ratio - distances[i]) / (distances[i + 1] - distances[i]);
        double volume = volumes[i] * (1 - t) + volumes[i + 1] * t;


        // temperature
        double T_j = Temp_B[BGeo];
        double T_i = Temp_A[AGeo];
        double Q_ij = ks * volume *  (T_j - T_i);
        atomicAdd(Q_A + AGeo, Q_ij);
        atomicAdd(Q_B + BGeo, -Q_ij);
        }
    }
